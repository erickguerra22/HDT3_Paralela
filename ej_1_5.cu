#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Modified by   : G. Erick - 21781
 Last modified : November 2024
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
   printf("Bloque: %d | Hilo: %d. Erick Guerra - 21781\n", blockIdx.x, threadIdx.x);
}

int main()
{
  hello<<<1,2048>>>();
  hipDeviceSynchronize(); //deprecated
  return 0;
}
