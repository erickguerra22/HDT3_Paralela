#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Modified by   : G. Erick - 21781
 Last modified : November 2024
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello2.cu -o hello2 -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello ()
{
   int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
                blockIdx.y * gridDim.x +
                blockIdx.x ) * blockDim.x * blockDim.y * blockDim.z +
                threadIdx.z *  blockDim.x * blockDim.y +
                threadIdx.y * blockDim.x +
                threadIdx.x;

//  Simplification of above
  //grid: 3D --- z,y,x: all dims and blockids
  //block: 1D -- x
//  int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
//               blockIdx.y * gridDim.x +
//               blockIdx.x ) * blockDim.x +
//               threadIdx.x;
   
    int maxThread = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    int maxID = maxThread - 1;

    if (myID == maxID)
    {
        printf ("Erick Guerra - 21781. MAX ID: %i\n", myID);
    }
}

int main ()
{
  dim3 g (10,10,10);
  dim3 b (100,10);
  hello <<< g, b >>> ();
  hipDeviceSynchronize ();
  //hipDeviceSynchronize();  //use instead, ^ is deprecated
  return 0;
}
